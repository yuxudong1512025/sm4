#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include"pch.h"
#include"sm4.h"

using namespace std;
#ifndef CHECK
#define  CHECK(call){\
	const hipError_t error = call;\
	if (error != hipSuccess) {\
		printf_s("Error: %s:%d, ", __FILE__, __LINE__);\
		printf_s("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(-10 * error);\
	}\
}
#endif

#ifndef GET_ULONG_BE
#define GET_ULONG_BE(n,b,i)                             \
{                                                       \
    (n) = ( (unsigned long) (b)[(i)    ] << 24 )        \
        | ( (unsigned long) (b)[(i) + 1] << 16 )        \
        | ( (unsigned long) (b)[(i) + 2] <<  8 )        \
        | ( (unsigned long) (b)[(i) + 3]       );       \
}
#endif

#ifndef PUT_ULONG_BE
#define PUT_ULONG_BE(n,b,i)                             \
{                                                       \
    (b)[(i)    ] = (unsigned char) ( (n) >> 24 );       \
    (b)[(i) + 1] = (unsigned char) ( (n) >> 16 );       \
    (b)[(i) + 2] = (unsigned char) ( (n) >>  8 );       \
    (b)[(i) + 3] = (unsigned char) ( (n)       );       \
}
#endif


#define  SHL(x,n) (((x) & 0xFFFFFFFF) << n)
#define ROTL(x,n) (SHL((x),n) | ((x) >> (32 - n)))
#define doublec(n) (n*n)


u8 SboxTable[256] = {
0xd6,0x90,0xe9,0xfe,0xcc,0xe1,0x3d,0xb7,0x16,0xb6,0x14,0xc2,0x28,0xfb,0x2c,0x05,
0x2b,0x67,0x9a,0x76,0x2a,0xbe,0x04,0xc3,0xaa,0x44,0x13,0x26,0x49,0x86,0x06,0x99,
0x9c,0x42,0x50,0xf4,0x91,0xef,0x98,0x7a,0x33,0x54,0x0b,0x43,0xed,0xcf,0xac,0x62,
0xe4,0xb3,0x1c,0xa9,0xc9,0x08,0xe8,0x95,0x80,0xdf,0x94,0xfa,0x75,0x8f,0x3f,0xa6,
0x47,0x07,0xa7,0xfc,0xf3,0x73,0x17,0xba,0x83,0x59,0x3c,0x19,0xe6,0x85,0x4f,0xa8,
0x68,0x6b,0x81,0xb2,0x71,0x64,0xda,0x8b,0xf8,0xeb,0x0f,0x4b,0x70,0x56,0x9d,0x35,
0x1e,0x24,0x0e,0x5e,0x63,0x58,0xd1,0xa2,0x25,0x22,0x7c,0x3b,0x01,0x21,0x78,0x87,
0xd4,0x00,0x46,0x57,0x9f,0xd3,0x27,0x52,0x4c,0x36,0x02,0xe7,0xa0,0xc4,0xc8,0x9e,
0xea,0xbf,0x8a,0xd2,0x40,0xc7,0x38,0xb5,0xa3,0xf7,0xf2,0xce,0xf9,0x61,0x15,0xa1,
0xe0,0xae,0x5d,0xa4,0x9b,0x34,0x1a,0x55,0xad,0x93,0x32,0x30,0xf5,0x8c,0xb1,0xe3,
0x1d,0xf6,0xe2,0x2e,0x82,0x66,0xca,0x60,0xc0,0x29,0x23,0xab,0x0d,0x53,0x4e,0x6f,
0xd5,0xdb,0x37,0x45,0xde,0xfd,0x8e,0x2f,0x03,0xff,0x6a,0x72,0x6d,0x6c,0x5b,0x51,
0x8d,0x1b,0xaf,0x92,0xbb,0xdd,0xbc,0x7f,0x11,0xd9,0x5c,0x41,0x1f,0x10,0x5a,0xd8,
0x0a,0xc1,0x31,0x88,0xa5,0xcd,0x7b,0xbd,0x2d,0x74,0xd0,0x12,0xb8,0xe5,0xb4,0xb0,
0x89,0x69,0x97,0x4a,0x0c,0x96,0x77,0x7e,0x65,0xb9,0xf1,0x09,0xc5,0x6e,0xc6,0x84,
0x18,0xf0,0x7d,0xec,0x3a,0xdc,0x4d,0x20,0x79,0xee,0x5f,0x3e,0xd7,0xcb,0x39,0x48
 };


/*(1024*2^14*256*1)*/
/*�޶�����ע����C31��һ���ֽ�*/
/*ciphertxt[2 * n]=C32^C33^C34, ciphertxt[2 * n + 1]=C35    */
__global__ void kernel(u32 *guessKey, u32 *maxSEI, u32 *ciphertxt, int countn,int i,u8 *Count,u8 *Sbox) {
	u32 ix = blockIdx.x*blockDim.x + threadIdx.x;
	u32 key = (i<<10)+ix;

	u32 MaxSei = 0;
	u32 ka;
	u8 a[4],b[4];
	u8 ans=0;

	for (int i = 0; i < countn; i++) {
		ka = key ^ ciphertxt[2 * i];
		PUT_ULONG_BE(ka, a, 0)
		b[0] = Sbox[a[0]];
		b[1] = Sbox[a[1]];
		b[2] = Sbox[a[2]];
		b[3] = Sbox[a[3]];
		GET_ULONG_BE(ka, b, 0)
		ans = b[0]^ b[3]^ (u8)(ka >> 6)^ (u8)(ka>> 14)^ (u8)(ka >> 22)^ (u8)(ciphertxt[2 * i + 1] >> 24);
		Count[ix * 256 + ans]++;
	}
	for (int i = 0; i < 256; i++)MaxSei += doublec(Count[ix * 256 + i]);

	guessKey[ix] = key;
	maxSEI[ix] = MaxSei;
}


__global__ void getMaxSEI(u32 *maxSEI, u32 *maxKey, u32 *SEIlist, u32 *KEYlist,int i) {//<<<(16384,1),(1024,1)>>>======<<<(256,1)(256,1)>>>
	const u32 tid = threadIdx.x;
	const u32 it = tid + blockIdx.x*blockDim.x;
	for (int stride = blockDim.x *gridDim.x / 2; stride > 0; stride = stride >> 1) {
		if (it < stride) {
			if (maxSEI[it] < maxSEI[it + stride]) {
				maxSEI[it] = maxSEI[it + stride];
				maxKey[it] = maxKey[it + stride];
			}
			__syncthreads();
		}
		__syncthreads();
	}

	if (it == 0) {
		SEIlist[i] = maxSEI[it];
		KEYlist[i] = maxKey[it];
	}
}

extern "C"
u32 getKey_Stream(u32 *ciphertxt0, int Countn, const u32 &trueKey) {
	//����豸��Ϣ��������
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf_s("using device %d : %s \n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));
	//���ò���
	int size_14 = 1 << 14;
	int size_8 = 1 << 8;
	int size_22 = 1 << 22;
	int nsize = size_22 * Countn * sizeof(u8);
	//	printf_s("Matrix size:nx %d ny %d\n", nx, ny);

	//����Ϊ������̲߳���gird=��256��65536����block=��256,1����threads
	dim3 grid(size_14, 1);
	dim3 block(size_8, 1);
	///////////////////////////////////////////////////////

	//����device�ڴ沢��host���ڴ濽����device�ϣ����룩
	u32 *cipher;

	
	CHECK(hipMalloc((void **)&cipher, Countn * 2 * sizeof(u32)));

	CHECK(hipMemcpy(cipher, ciphertxt0, Countn * 2 * sizeof(u32), hipMemcpyHostToDevice));
	//����device�ڴ���Ϊ����ռ䣨�����
	u32 *maxSEI;
	u32 *maxKey;
	u8 *Count,*SBOX;
	CHECK(hipMalloc((void **)&maxSEI, nsize));
	CHECK(hipMalloc((void **)&maxKey, nsize));
	CHECK(hipMalloc((void **)&Count,size_22*256*sizeof(u8)));
	CHECK(hipMalloc((void **)&SBOX,  256 * sizeof(u8)));

	CHECK(hipMemcpy(SBOX,SboxTable, 256 * sizeof(u8),hipMemcpyHostToDevice));
	//����ѭ�������ſռ�
	u32 *ansSEI, *ansKEY;
	CHECK(hipMalloc((void **)&ansSEI, 1024*sizeof(u32)));
	CHECK(hipMalloc((void **)&ansKEY, 1024 * sizeof(u32)));

	for (int i = 0; i < 1024; i++) {
		CHECK(hipMemset(Count,0,sizeof(Count)));
		//���ú˺�������SEI
		kernel << <grid, block >> > (maxKey,maxSEI,cipher,Countn,i,Count,SBOX);
		CHECK(hipDeviceSynchronize());//���cuda�豸ͬ�����
		//���ú˺�����SEI����key
		getMaxSEI << <grid, block >> > (maxSEI,maxKey,ansSEI,ansKEY,i);
		CHECK(hipDeviceSynchronize());//���cuda�豸ͬ�����
	}
	
	u32 *ansS, *ansK, aS=0, aK;
	ansS = (u32*)malloc(1024 * sizeof(u32));
	ansK = (u32*)malloc(1024 * sizeof(u32));

	//�������������host
	CHECK(hipMemcpy(ansS, ansSEI, 1024 * sizeof(u32), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(ansK, ansKEY, 1024 * sizeof(u32), hipMemcpyDeviceToHost));

	for (int i = 0; i < 1024; i++) {
		if (aS < ansS[i]) {
			aS = ansS[i];
			aK = ansK[i];
		}
	}

	free(ansS);
	free(ansK);
	printf("%d,%x", aS, aK);




	//�ͷ�ռ�ÿռ�
	CHECK(hipFree(maxSEI));
	CHECK(hipFree(maxKey));
	CHECK(hipFree(cipher));
	CHECK(hipFree(Count));
	CHECK(hipFree(ansSEI));
	CHECK(hipFree(SBOX));
	CHECK(hipFree(ansKEY));
	return aK;
}